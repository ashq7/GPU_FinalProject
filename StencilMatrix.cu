#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <ostream>

using namespace std;

//#define N 64 -what was N? I think N+2*Radius = DSIZE
#define DSIZE 512
#define RADIUS 3
#define BLOCK_SIZE 32


__global__ void stencil_2d(int *in, int *out) {

	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];
	int gindex_x = threadIdx.x + blockIdx.x * blockDim.x; 
	int lindex_x = threadIdx.x + RADIUS;
	int gindex_y = threadIdx.y + blockIdx.y * blockDim.y;
	int lindex_y = threadIdx.y + RADIUS;

	// Read input elements into shared memory
	//int size = N + 2 * RADIUS; //becomes DSIZE
	temp[lindex_x][lindex_y] = in[gindex_y + DSIZE * gindex_x]; 

	if (threadIdx.x < RADIUS) {
		temp[lindex_x-RADIUS][lindex_y]=in[gindex_y + DSIZE * (gindex_x - RADIUS)];
		temp[lindex_x + BLOCK_SIZE][lindex_y] = in[gindex_y + DSIZE * (gindex_x + BLOCK_SIZE)]; 
	}

	if (threadIdx.y < RADIUS ) {
		temp[lindex_x][lindex_y-RADIUS]=in[(gindex_y - RADIUS)+ DSIZE * gindex_x];
		temp[lindex_x][lindex_y + BLOCK_SIZE] = in[gindex_y + BLOCK_SIZE + DSIZE * gindex_x];
	}


	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++){
		__syncthreads(); //makes sure we have access to everything in temp accessed across multiple threads
		result += temp[lindex_x + offset][lindex_y];
		//avoid double-counting 
		if(offset!=0){
			result += temp[lindex_x][lindex_y + offset];
		}
	}

	// Store the result
	out[gindex_y+DSIZE*gindex_x] = result;
}

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const int *A, const int *B, int *C, int size) {
  //FIXME:
  // i iterates over rows of matrix A
  for (int i = 0; i<size; i++){
    // j iterates over columns of matrix B
    for (int j = 0; j<size; j++){
        int temp = 0;
        // k indexes which item in the ith row of A and jth column of B we are multiplying
        for (int k = 0; k<size; k++){
            //i is analagous to idx, j to idy, size to n
            temp += A[i * size + k] * B [k * size + j];
        }
    C[i*size + j]= temp;
    }
  }
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const int *A, const int *B, int *C, int size) {

    //FIXME:
    // create thread x index
    // create thread y index
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        int temp = 0;
        for (int i = 0; i < size; i++){
            //FIXME : Add dot product of row and column
            temp += A [idx * size +idy] * B [idy * size +idx];
        }
        C[idx*size+idy] = temp;                    
    }

}

// Error Checking for stencil
    int error_stencil (*int stencilled, *int original){
        for (int i = 0; i < DSIZE; ++i) {
            for (int j = 0; j < DSIZE; ++j) {

                if (i < RADIUS || DSIZE-i<= RADIUS) {
                    if (stencilled[j+i*DSIZE] != original) {
                        printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(DSIZE)], 1);
                        return -1;
                    }
                }
                else if (j < RADIUS || DSIZE-j<= RADIUS) {
                    if (stencilled[j+i*(DSIZE)] != original) {
                        printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(DSIZE)], 1);
                        return -1;
                    }
                }		 
                else { // EDIT- wrong!
                    if (stencilled[j+i*(DSIZE)] != 1 + 4 * RADIUS) {
                        printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(DSIZE)], 1 + 4*RADIUS);
                        return -1;
                    }
                }
            }
	    }
        return 0;
    }

// error checking macro from matrix multiplication
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)


void fill_ints(int *x, int n) {
   // Store the result
   // https://en.cppreference.com/w/cpp/algorithm/fill_n
   fill_n(x, n, 1);
   //takes in matrix, starts at pointer and fills subsequent n with value (1 here)
}


int main(void) {

    int *h_A, *h_A_stencilled, *h_B, *h_B_stencilled, *h_C; //host copies
    int *d_A, *d_A_stencilled, *d_B, *d_B_stencilled, *d_C; //device copies

    //Alloc space for host copies 
    int size = (DSIZE)*(DSIZE) * sizeof(int);
    h_A = (int*)malloc(size);
    h_A_stencilled = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_B_stencilled = (int*)malloc(size);
    h_C = (int*)malloc(size);

    //initialize host values
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = (rand() % 10);
        h_B[i] = (rand() % 10);
        h_A_stencilled[i]=0;
        h_B_stencilled[i]=0;
        h_C[i] = 0;
    }

    // Allocate device memory 
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_A_stencilled, size);
    hipMalloc((void **)&d_B_stencilled, size);
    hipMalloc((void **)&d_C, size);
    cudaCheckErrors("After Memory Allocation");

    // Copy from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_A_stencilled, h_A_stencilled, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B_stencilled, h_B_stencilled, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	// Launch stencil_2d() kernel on GPU
	int gridSize = DSIZE/BLOCK_SIZE; //from Asignment 2 mult_matrix.cu
	dim3 grid(gridSize, gridSize);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	
	stencil_2d<<<grid,block>>>(d_A + RADIUS*(DSIZE) + RADIUS , d_A_stencilled + RADIUS*(DSIZE) + RADIUS); //QUESTION: confused how the plus works?
	stencil_2d<<<grid,block>>>(d_B + RADIUS*(DSIZE) + RADIUS , d_B_stencilled + RADIUS*(DSIZE) + RADIUS);

    //Launch matrix_mul kernel on GPU
    matrix_mul_gpu<<<grid,block>>>(d_A_stencilled, d_B_stencilled, d_C, size);


	// Copy result back to host
	hipMemcpy(h_A_stencilled, d_A_stencilled, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_B_stencilled, d_B_stencilled, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	
	

	// Free memory 
    free(h_A);
    free(h_B);
    free(h_A_stencilled);
    free(h_B_stencilled);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_A_stencilled);
    hipFree(d_B_stencilled);
    hipFree(d_C);

}


